
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t sumaCuda(int *d, const int *a, const int *b, const int *c);

//Kernel
__global__ void sumaKernel(int *d, const int *a, const int *b, const int *c)
{
    *d = *a + *b + *c;
}

int main()
{
	//Diego-Edgar Gracia Pe�a
	const int a = 4, b = 7, c = 3;
	int d = 0;

    // Add vectors in parallel.
    hipError_t cudaStatus = sumaCuda(&d, &a, &b, &c);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Fallo en sumaCuda");
        return 1;
    }

    printf("%d + %d + %d = %d\n", a, b, c, d);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
	printf("Codificado por Diego-Edgar Gracia Pe�a");
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t sumaCuda(int *d, const int *a, const int *b, const int *c)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
	int *dev_d = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "�fallo en cudaSetDevice!  �Seguro que tienes una GPU con capacidad para CUDA?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_d, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_c, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {//DEGP
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_c, c, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
    sumaKernel<<<1, 1>>>(dev_d, dev_a, dev_b, dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "fallo en el lanzamiento de sumaKernel: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(d, dev_d, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_d);
    hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
    
    return cudaStatus;
}
